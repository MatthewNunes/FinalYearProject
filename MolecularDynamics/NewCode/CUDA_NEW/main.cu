#include "hip/hip_runtime.h"
/*
FILE  main.c

DOC   This program simulates a three-dimensional Lennard-Jones fluid.

LANG  C plus MPI message passing.

HIS   1) Parallel version originally written by University of
HIS      Southampton.
HIS   2) Adopted to run on the Intel iPSC/2 computer at Daresbury
HIS      Laboratory, and placed in the public domain through the
HIS      CCP5 programme.
HIS   3) Adopted to run on the Intel Paragon and enhanced by David W.
HIS      Walker at Oak Ridge National Laboratory, Tennessess, USA
HIS   4) Converted to use MPI message passing library by David W. Walker
HIS      at University of Wales Cardiff in July 1997.
HIS   5) Converted from Fortran to C in November 1997.

*/

#include <stdio.h>
#include <stdlib.h>
#include "moldyn.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "matthew_CUDA.h"

#define BLOCK_WIDTH 24


long unsigned int get_tick()
{
   struct timespec ts;
   if (clock_gettime(CLOCK_MONOTONIC, &ts) != 0) return (0);
   return ts.tv_sec*(long int)1000 + ts.tv_nsec / (long int) 1000000;
}

int main ( int argc, char *argv[])
{
   float sigma, rcut, dt, eqtemp, dens, boxlx, boxly, boxlz, sfx, sfy, sfz, sr6, vrcut, dvrcut, dvrc12, freex; 
   int nstep, nequil, iscale, nc, mx, my, mz, iprint;
   float *rx, *ry, *rz, *vx, *vy, *vz, *fx, *fy, *fz, *potentialPointer, *virialPointer, *virialArray, *potentialArray, *virialArrayTemp, *potentialArrayTemp;
   float *d_rx, *d_ry, *d_rz, *d_fx, *d_fy, *d_fz, *d_potential, *d_virial, *d_virialArray, *d_potentialArray;
   int *d_head, *d_list;
   float ace, acv, ack, acp, acesq, acvsq, acksq, acpsq, vg, kg, wg;
   int   *head, *list;
   int   natoms=0;
   int ierror;
   int jstart, step, itemp;
   float potential, virial, kinetic;
   float tmpx;
   int i, icell;
   hipSetDevice(2);
   ierror = input_parameters (&sigma, &rcut, &dt, &eqtemp, &dens, &boxlx, &boxly, &boxlz, &sfx, &sfy, &sfz, &sr6, &vrcut, &dvrcut, &dvrc12, &freex, &nstep, &nequil, &iscale, &nc, &natoms, &mx, &my, &mz, &iprint);
   //printf ("\nReturned from input_parameters, natoms = %d\n", natoms);
  // CUDA_CHECK_RETURN(hipSetDevice(1));
   //printf("\nmx = %d, my = %d, mz = %d\n",mx,my,mz);
   rx = (float *)malloc(2*natoms*sizeof(float));
   ry = (float *)malloc(2*natoms*sizeof(float));
   rz = (float *)malloc(2*natoms*sizeof(float));
   vx = (float *)malloc(natoms*sizeof(float));
   vy = (float *)malloc(natoms*sizeof(float));
   vz = (float *)malloc(natoms*sizeof(float));
   fx = (float *)malloc(natoms*sizeof(float));
   fy = (float *)malloc(natoms*sizeof(float));
   fz = (float *)malloc(natoms*sizeof(float));
   list = (int *)malloc(2*natoms*sizeof(int));
   head= (int *)malloc((mx+2)*(my+2)*(mz+2)*sizeof(int));
   virialPointer = (float *)malloc(sizeof(float));
   potentialPointer = (float *)malloc(sizeof(float));

   

   
  // printf ("\nFinished allocating memory\n");

   initialise_particles (rx, ry, rz, vx, vy, vz, nc);
 //  printf ("\nReturned from initialise_particles\n");

   loop_initialise(&ace, &acv, &ack, &acp, &acesq, &acvsq, &acksq, &acpsq, sigma, rcut, dt);
//   printf ("\nReturned from loop_initialise\n");

//   output_particles(rx,ry,rz,vx,vy,vz,fx,fy,fz,0);
      movout (rx, ry, rz, vx, vy, vz, sfx, sfy, sfz, head, list, mx, my, mz, natoms);

   int natomsSqrt = ceil(sqrt(natoms));
   int numBlocks = ceil(natomsSqrt/(float)(BLOCK_WIDTH));
   virialArrayTemp = (float *)malloc((numBlocks * numBlocks)* sizeof(float));
   potentialArrayTemp = (float *)malloc((numBlocks * numBlocks) * sizeof(float));
   virialArray = (float *)malloc((numBlocks * numBlocks)* sizeof(float));
   potentialArray = (float *)malloc((numBlocks * numBlocks) * sizeof(float));
   int index;
   for (index = 0; index < (numBlocks * numBlocks); index++)
   {
      virialArray[index] = (float)0;
      potentialArray[index] = (float)0;
   }
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_rx, 2*natoms * sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_ry, 2*natoms * sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_rz, 2*natoms * sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_fx, natoms * sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_fy, natoms * sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_fz, natoms * sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_head, (mx+2) * (my+2) * (mz+2) * sizeof(int)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_list, 2 * natoms * sizeof(int)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_potential, sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_virial, sizeof(float)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_virialArray, sizeof(float) * (numBlocks * numBlocks)));
   CUDA_CHECK_RETURN(hipMalloc((void **) &d_potentialArray, sizeof(float) * (numBlocks * numBlocks)));

      //*potentialPointer = (float)0;
      //*virialPointer = (float)0;
      CUDA_CHECK_RETURN(hipMemcpy(d_rx, rx, 2*natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_ry, ry, 2*natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_rz, rz, 2*natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_fx, fx, natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_fy, fy, natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_fz, fz, natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_head, head, (mx+2) * (my+2) * (mz+2) * sizeof(int), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_list, list, 2*natoms * sizeof(int), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_potential, potentialPointer, sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_virial, virialPointer, sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_virialArray, virialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_potentialArray, potentialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyHostToDevice));
 //  printf ("\nReturned from movout\n");
   //   check_cells(rx, ry, rz, head, list, mx, my, mz, natoms,0,0);

      long double elapsedTime = (float)0;
      long unsigned int startTime;
      long unsigned int endTime;
      startTime = get_tick();
      printf("numBlocks: %d\n", numBlocks);
      printf("BLOCK_WIDTH: %d\n", BLOCK_WIDTH);
      //copySigma(&sigma);
      //printf("main sigma: %f\n", sigma);
      /**
      int stepTemp = 0;
      copySigma(&sigma);
      copyRcut(&rcut);
      copyVrcut(&vrcut);
      copyDvrc12(&dvrc12);
      copyDvrcut(&dvrcut);
      copyMx(&mx);
      copyMy(&my);
      copyMz(&mz);
      copyNatoms(&natoms);
      copyStep(&stepTemp);
      copySfx(&sfx);
      copySfy(&sfy);
      copySfz(&sfz);
      */
      float *ppTemp = (float *) malloc(sizeof(float));
      float *vvTemp = (float *) malloc(sizeof(float));
      dim3 numberOfBlocks(numBlocks, numBlocks);
      dim3 numberOfThreads(BLOCK_WIDTH, BLOCK_WIDTH);
      force<<<numberOfBlocks, numberOfThreads>>>(d_virialArray, d_potentialArray, d_potential, d_virial, d_rx, d_ry, d_rz, d_fx, d_fy, d_fz, sigma, rcut, vrcut, dvrc12, dvrcut, d_head, d_list, mx, my, mz, natoms, step, sfx, sfy, sfz);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
      CUDA_CHECK_RETURN(hipDeviceSynchronize());
      endTime = get_tick();   
      elapsedTime += endTime - startTime;
      CUDA_CHECK_RETURN(hipMemcpy(fx, d_fx, natoms * sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(fy, d_fy, natoms * sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(fz, d_fz, natoms * sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(potentialArrayTemp, d_potentialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(virialArrayTemp, d_virialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyDeviceToHost));
     // CUDA_CHECK_RETURN(hipMemcpy(ppTemp, d_potential, sizeof(float), hipMemcpyDeviceToHost));
     // CUDA_CHECK_RETURN(hipMemcpy(vvTemp, d_virial, sizeof(float), hipMemcpyDeviceToHost));
      //hipMemcpy(virialTest, d_virialArray, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);
      //hipMemcpy(potentialTest, d_potentialArray, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);
      startTime = get_tick();
      //numBlocks +=1;
      //finalResult<<<numBlocks, 1>>>(d_potentialArray, d_virialArray, d_potential, d_virial, numBlocks);
      //CUDA_CHECK_RETURN(hipPeekAtLastError());
      //CUDA_CHECK_RETURN(hipDeviceSynchronize());
      endTime = get_tick();
      elapsedTime += endTime - startTime;
      //hipMemcpy(potentialArrayTemp, d_potentialArray, sizeof(float) * (numBlocks + 1), hipMemcpyDeviceToHost);
      //hipMemcpy(virialArrayTemp, d_virialArray, sizeof(float) * (numBlocks + 1), hipMemcpyDeviceToHost);
      int tempInd = 0;
      virial = 0.0;
      potential = 0.0;
      for (tempInd = 0; tempInd < (numBlocks * numBlocks); tempInd++)
      {
         virial += virialArrayTemp[tempInd];
         potential += potentialArrayTemp[tempInd];
      }
      //printf("virial: %f\n", virial);
      //printf("potential: %f\n", potential);
     // virial = virialArrayTemp[0] * (48.0/3.0);
      //potential = potentialArrayTemp[0] * 4.0;
      virial *= 48.0/3.0;
      potential *= 4.0;
      
  // printf ("\nReturned from force: potential = %f, virial = %f, kinetic = %f\n",potential, virial, kinetic);
//   output_particles(rx,ry,rz,vx,vy,vz,fx,fy,fz,0);


   for(step=1;step<=nstep;step++){
     // if(step>=85)printf ("\nStarted step %d\n",step);
      movea (rx, ry, rz, vx, vy, vz, fx, fy, fz, dt, natoms);
//      check_cells(rx, ry, rz, head, list, mx, my, mz, natoms,step,step);
    //  if(step>85)printf ("\nReturned from movea\n");
      movout (rx, ry, rz, vx, vy, vz, sfx, sfy, sfz, head, list, mx, my, mz, natoms);

      CUDA_CHECK_RETURN(hipMemcpy(d_rx, rx, 2*natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_ry, ry, 2*natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_rz, rz, 2*natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_fx, fx, natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_fy, fy, natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_fz, fz, natoms * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_head, head, (mx+2)*(my+2)*(mz+2) * sizeof(int), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_list, list, 2*natoms * sizeof(int), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_virialArray, virialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(d_potentialArray, potentialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyHostToDevice));

   //  if(step>85) printf ("\nReturned from movout\n");
  //    check_cells(rx, ry, rz, head, list, mx, my, mz, natoms,step,step);
      startTime = get_tick();
      //numBlocks -= 1;
     // int stepTemp = 0;
      /**
      copySigma(&sigma);
      copyRcut(&rcut);
      copyVrcut(&vrcut);
      copyDvrc12(&dvrc12);
      copyDvrcut(&dvrcut);
      copyMx(&mx);
      copyMy(&my);
      copyMz(&mz);
      copyNatoms(&natoms);
      copyStep(&stepTemp);
      copySfx(&sfx);
      copySfy(&sfy);
      copySfz(&sfz);
      */
      //printf("main sigma: %f\n", sigma);
      force<<<numberOfBlocks, numberOfThreads>>>(d_virialArray, d_potentialArray, d_potential, d_virial, d_rx, d_ry, d_rz, d_fx, d_fy, d_fz, sigma, rcut, vrcut, dvrc12, dvrcut, d_head, d_list, mx, my, mz, natoms, step, sfx, sfy, sfz);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
      CUDA_CHECK_RETURN(hipDeviceSynchronize());
      endTime = get_tick();
      elapsedTime += endTime - startTime;
      CUDA_CHECK_RETURN(hipMemcpy(fx, d_fx, natoms * sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(fy, d_fy, natoms * sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(fz, d_fz, natoms * sizeof(float), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(potentialArrayTemp, d_potentialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyDeviceToHost));
      CUDA_CHECK_RETURN(hipMemcpy(virialArrayTemp, d_virialArray, sizeof(float) * (numBlocks * numBlocks), hipMemcpyDeviceToHost));
      //hipMemcpy(virialTest, d_virialArray, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);
      //hipMemcpy(potentialTest, d_potentialArray, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);
      startTime = get_tick();
      //numBlocks +=1;
      //finalResult<<<numBlocks, 1>>>(d_potentialArray, d_virialArray, d_potential, d_virial, numBlocks);
      //CUDA_CHECK_RETURN(hipPeekAtLastError());
      //CUDA_CHECK_RETURN(hipDeviceSynchronize());
      endTime = get_tick();
      elapsedTime += endTime - startTime;
      //hipMemcpy(potentialArrayTemp, d_potentialArray, sizeof(float) * (numBlocks + 1), hipMemcpyDeviceToHost);
      //hipMemcpy(virialArrayTemp, d_virialArray, sizeof(float) * (numBlocks + 1), hipMemcpyDeviceToHost);
      tempInd = 0;
      virial = 0.0;
      potential = 0.0;
      for (tempInd = 0; tempInd < (numBlocks * numBlocks); tempInd++)
      {
         virial += virialArrayTemp[tempInd];
         potential += potentialArrayTemp[tempInd];
      }
     // virial = virialArrayTemp[0] * (48.0/3.0);
      //potential = potentialArrayTemp[0] * 4.0;
     // printf("virial: %f\n", virial);
      //printf("potential: %f\n", potential);
      virial *= 48.0/3.0;
      potential *= 4.0;
//      if(step>85)printf ("\nReturned from force: potential = %f, virial = %f, kinetic = %f\n",potential, virial, kinetic);
 //     fflush(stdout);
      moveb (&kinetic, vx, vy, vz, fx, fy, fz, dt, natoms);
 //     check_cells(rx, ry, rz, head, list, mx, my, mz, natoms,step,step);
  // if(step>85)   printf ("\nReturned from moveb: potential = %f, virial = %f, kinetic = %f\n",potential, virial, kinetic);
      sum_energies (potential, kinetic, virial, &vg, &wg, &kg);
      hloop (kinetic, step, vg, wg, kg, freex, dens, sigma, eqtemp, &tmpx, &ace, &acv, &ack, &acp, &acesq, &acvsq, &acksq, &acpsq, vx, vy, vz, iscale, iprint, nequil, natoms);
   }

   tidyup (ace, ack, acv, acp, acesq, acksq, acvsq, acpsq, nstep, nequil);
   elapsedTime = elapsedTime / (float) 1000;
   printf("\n%Lf seconds have elapsed\n", elapsedTime);
   hipFree(d_fx);
   hipFree(d_fy);
   hipFree(d_fz);
   hipFree(d_rx);
   hipFree(d_ry);
   hipFree(d_rz);
   hipFree(d_head);
   hipFree(d_list);
   hipFree(virialArray);
   hipFree(potentialArray);
   hipFree(virialPointer);
   hipFree(potentialPointer);
   return 0;
}
