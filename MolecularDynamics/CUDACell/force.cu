#include "hip/hip_runtime.h"

#include <math.h>
#include <stdio.h>

#include "moldyn.h"

#define BLOCK_SIZE 512

__global__ void force (float *potentialArray, float *virialArray, float *pval, float *vval, float *rx, float *ry, float *rz, float *fx, float *fy, float *fz, float sigma, float rcut, float vrcut, float dvrc12, float dvrcut, int *head, int *list, int *map, int mx, int my, int mz, int natoms, int step)
{
   float sigsq, rcutsq;
   float rxi, ryi, rzi, fxi, fyi, fzi;
   float rxij, ryij, rzij, rijsq;
   float rij, sr2, sr6, vij, wij, fij, fxij, fyij, fzij;
   float potential, virial;
   int i, icell, j, jcell, nabor;
   int xi, yi, zi, ix, jx, kx, icount, xcell, ycell, zcell;
   float valv, valp;

   sigsq  = sigma*sigma;
   rcutsq = rcut*rcut;

//   for(i=0;i<natoms;++i){
//      *(fx+i) = 0.0;
//      *(fy+i) = 0.0;
//      *(fz+i) = 0.0;
//   }

   potential = 0.0;
   virial    = 0.0;
   valv = 0.0;
   valp = 0.0;
   
   icount = 0;
   int element = blockDim.x * blockIdx.x + threadIdx.x;

   potentialArray[element] = 0.0;
   virialArray[element] = 0.0;
   if(element < ((mx+2) * (my + 2) * (mz + 2)))
   {

      xi = element%(mx+2);
      yi = (element/(mx+2))%(my+2);
      zi = element/((mx+2)*(my+2));
      if(((xi>0) && (xi <(mx+1)))&&((yi>0) && (yi<(my+1)))&&((zi>0) && (zi<(mz+1))))
      {
        i = head[element];
        while (i>=0) 
        {
          rxi = rx[i];
          ryi = ry[i];
          rzi = rz[i];
//	 printf("Particle %5d, (xi,yi,zi) = %d,%d,%d, icel = %5d\n",i,xi,yi,zi,icell);
//               fxi = fx[i];
 //              fyi = fy[i];
  //             fzi = fz[i];
          fxi = fyi = fzi = 0.0;

          j = head[element];
          while (j>=0) 
          {
            rxij = rxi - rx[j];
            ryij = ryi - ry[j];
            rzij = rzi - rz[j];
            rijsq = rxij*rxij + ryij*ryij + rzij*rzij;
            if ((rijsq < rcutsq) && (j!=i)) 
            {
                    //START FORCE IJ
	               //force_ij(rijsq, rxij, ryij, rzij, sigsq, vrcut, dvrc12, rcut, dvrcut, &vij, &wij, &fxij, &fyij, &fzij);
                
              rij = (float) sqrt ((double)rijsq);
              sr2 = sigsq/rijsq;
              sr6 = sr2*sr2*sr2;
              vij = sr6*(sr6-1.0) - vrcut - dvrc12*(rij-rcut);
              wij = sr6*(sr6-0.5) + dvrcut*rij;
              fij = wij/rijsq;
              fxij = fij*rxij;
              fyij = fij*ryij;
              fzij = fij*rzij;
		          //END FORCE IJ
              vij *= 0.5;
		          wij *= 0.5;
		          valp += vij;
		          valv += wij;
//                       potential += 0.5*vij;
//                       virial    += 0.5*wij;
              fxi+= fxij;
              fyi+= fyij;
              fzi+= fzij;
            }           
            j = list[j];
          }
	        icount++;
//	      printf("\nCell %d at (%d,%d,%d) interacts with cells: ",icell,xi,yi,zi);
          for (ix=-1;ix<=1;ix++)
            for (jx=-1;jx<=1;jx++)
              for (kx=-1;kx<=1;kx++)
              {
                xcell = ix+xi;
                ycell = jx+yi;
                zcell = kx+zi;
                jcell = xcell + (mx+2)*(ycell+(my+2)*zcell);
			//       printf("%d (%d,%d,%d); ",jcell,xcell,ycell,zcell);
		            if(element!=jcell) 
                {
                  j = head[jcell];
                  while (j>=0) 
                  {
                    rxij = rxi - rx[j];
                    ryij = ryi - ry[j];
                    rzij = rzi - rz[j];
                    rijsq = rxij*rxij + ryij*ryij + rzij*rzij;
                    if (rijsq < rcutsq) 
                    {
                      //START FORCE IJ
                      rij = (float) sqrt ((double)rijsq);
                      sr2 = sigsq/rijsq;
                      sr6 = sr2*sr2*sr2;
                      vij = sr6*(sr6-1.0) - vrcut - dvrc12*(rij-rcut);
                      wij = sr6*(sr6-0.5) + dvrcut*rij;
                      fij = wij/rijsq;
                      fxij = fij*rxij;
                      fyij = fij*ryij;
                      fzij = fij*rzij;
                      //END FORCE IJ
                      wij *= 0.5;
                      vij *= 0.5;
				              valp += vij;
				              valv += wij;
                      fxi += fxij;
                      fyi += fyij;
                      fzi += fzij;
                    }
                    j = list[j];
			            }
                }		          
              }  
          *(fx+i) = 48.0*fxi;
          *(fy+i) = 48.0*fyi;
          *(fz+i) = 48.0*fzi;
          i = list[i];  
          //printf("valp: %f from element: %d\n", valp, element);
	        potential += valp;
	        virial += valv;
	        valp = valv = 0.0;           
	      }//While loop (current cell under consideration)
      }//if statement checking that cell's coordinates are within range
    }
    potentialArray[element] = potential;
    virialArray[element] = virial;
    //if statement over all cells
  // if(icount!=natoms) printf("\nProcessed %d particles in force routine instead of %d",icount,natoms);
  // potential *= 4.0;
  // virial    *= 48.0/3.0;
  // *pval = potential;
  // *vval = virial;

//   for (i=0;i<natoms;++i) {
//      *(fx+i) *= 48.0;
//      *(fy+i) *= 48.0;
//      *(fz+i) *= 48.0;
//   }
}
