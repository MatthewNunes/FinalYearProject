#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define WIDTH 500
#define BLOCK_WIDTH 16

__global__ 
void matMulKernel(float *d_M, float *d_N, float *d_P, int Width)
{ 
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    int k;
    if ((Row<Width)&&(Col<Width)){
        float Pvalue = 0.0;
        for(k=0;k<Width;k++)
            Pvalue += d_M[Row*Width+k]*d_N[k*Width+Col];
        d_P[Row*Width+Col] = Pvalue;
    }
}

void matMulDevice(float *h_M, float *h_N, float *h_P, int Width)
{
    struct timeval tim;
    int size = Width * Width * sizeof(float); 
    float *d_M, *d_N, *d_P;
// Step 1: Allocate and Load M, N to device memory 
    hipMalloc((void **)&d_M, size);
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_N, size);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
// Step 2: Allocate P on the device
    hipMalloc((void **)&d_P, size);
// Step 3a: Set up execution configuration
   int numBlocks = ceil(Width/(float)BLOCK_WIDTH);
   dim3 dimGrid(numBlocks,numBlocks);
   dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
// Step 3b: Launch the device computation threads!
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  // gettimeofday(&tim, NULL);
   hipEventRecord(start, 0);
   //double t1=tim.tv_sec+(tim.tv_usec/1000000.0);
   matMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, Width);
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   float elapsedTime;
   
   hipEventElapsedTime(&elapsedTime, start, stop);
   elapsedTime = elapsedTime / (float) 1000000;
   //gettimeofday(&tim, NULL);
   //double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
// Step 4: Copy back result, and free memory on device
   hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
   hipFree(d_M); hipFree(d_N); hipFree(d_P);
   printf("%.6f seconds elapsed\n", elapsedTime);
}

int main()
{
    float *h_M, *h_N, *h_P;
    int i, n = WIDTH, size=sizeof(float)*n*n;
    h_P = (float *)malloc(size);
    h_M = (float *)malloc(size);
    h_N = (float *)malloc(size);
    for(i=0;i<n*n;i++)
    {
        *(h_M+i)=(float)i; 
        *(h_N+i)=(float)i;
    }
    matMulDevice(h_M,h_N,h_P,n);
}
