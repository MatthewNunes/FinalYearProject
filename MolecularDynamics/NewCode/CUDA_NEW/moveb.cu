#include "hip/hip_runtime.h"
#include "moldyn.h"

__global__ void moveb (float *kineticArray, float *vx, float *vy, float *vz, float *fx, float *fy, float *fz, float dt, int natoms)
{
   float dt2;
   int i;
   int element = blockDim.x * blockIdx.x + threadIdx.x;
   float privateKinetic = 0.0;
   dt2 = dt*0.5;
  // *kinetic = 0.0;
   __shared__ float kineticSum[BLOCK_WIDTH];
   kineticSum[threadIdx.x] = 0.0
   if(element < natoms)
   {
      vx[element] = vx[element] + dt2*fx[element];
      vy[element] = vy[element] + dt2*fy[element];
      vz[element] = vz[element] + dt2*fz[element];
      kinetic += vx[element]*vx[element] + vy[element]*vy[element] + vz[element]*vz[element];
   }
   int stride;
   for (stride = blockDim.x/2; stride > 0; stride >>=1)
   {
      kineticSum[threadIdx.x] += kineticSum[threadIdx.x + stride];
   }
   if (threadIdx.x == 0)
   {   
      kineticArray[blockIdx.x] = kineticSum[0];
   }
}

__global__ void moveb(float *kineticArray, float *kinetic)
{
   int stride; 
   
   for (stride = blockDim.x/2; stride>0; stride>>=1)
   {

   }
}
