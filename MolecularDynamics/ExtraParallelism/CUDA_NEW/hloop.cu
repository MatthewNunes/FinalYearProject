#include "hip/hip_runtime.h"
#include "moldyn.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matthew_CUDA.h"
void hloop (float kinetic, int step, float vg, float wg, float kg, float freex, float dens, float sigma, float eqtemp, float *tmpx, float *ace, float *acv, float *ack, float *acp, float *acesq, float *acvsq, float *acksq, float *acpsq, float *vx, float *vy, float *vz, int iscale, int iprint, int nequil, int natoms, int block_width, float *d_vx, float *d_vy, float *d_vz)
{
   float e, en, vn, kn, pres;
  // float *d_tmpx;
   e = kg + vg;
   en = e/(float)natoms;
   vn = vg/(float)natoms;
   kn = kg/(float)natoms;
   *tmpx = 2.0*kg/freex;
   pres = dens*(*tmpx) + wg;
   pres = pres*sigma*sigma*sigma;
   
   if (step>nequil) {
      *ace += en;
      *acv += vn;
      *ack += kn;
      *acp += pres;
      *acesq += en*en;
      *acvsq += vn*vn;
      *acksq += kn*kn;
      *acpsq += pres*pres;
   }

/* If still equilibrating call subroutine to scale velocities */
   //CUDA_CHECK_RETURN(hipMalloc((void **) &d_tmpx, sizeof(float)));
   //CUDA_CHECK_RETURN(hipMemcpy(d_tmpx, tmpx, sizeof(float), hipMemcpyHostToDevice));
   int numBlocks2 = ceil(natoms/ (float) BLOCK_WIDTH2);
   if (nequil > step) 
   {
      scalet<<<numBlocks2, BLOCK_WIDTH2>>> (d_vx, d_vy, d_vz, kinetic, eqtemp, *tmpx, iscale, natoms, step);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
      CUDA_CHECK_RETURN(hipDeviceSynchronize());
      //CUDA_CHECK_RETURN(hipMemcpy(tmpx, d_tmpx, sizeof(float), hipMemcpyDeviceToHost));
   }
/* Optionally print information */
      if (step%iprint == 0)
         printf("\n%8d%12.6f%12.6f%12.6f%12.6f%12.6f",step, en, kn, vn,
                 pres, *tmpx);
}