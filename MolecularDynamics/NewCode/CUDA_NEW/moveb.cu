#include "hip/hip_runtime.h"
#include "moldyn.h"
#include <hip/hip_runtime.h>

__global__ void moveb (float *kineticArray, float *vx, float *vy, float *vz, float *fx, float *fy, float *fz, float dt, int natoms)
{
   float dt2;
   int element = blockDim.x * blockIdx.x + threadIdx.x;
   dt2 = dt*0.5;
  // *kinetic = 0.0;
   __shared__ float kineticSum[BLOCK_WIDTH];
   kineticSum[threadIdx.x] = 0.0;
   if(element < natoms)
   {
      vx[element] = vx[element] + dt2*fx[element];
      vy[element] = vy[element] + dt2*fy[element];
      vz[element] = vz[element] + dt2*fz[element];
      kineticSum[threadIdx.x] += vx[element]*vx[element] + vy[element]*vy[element] + vz[element]*vz[element];
   }
   int stride;
   for (stride = blockDim.x/2; stride > 0; stride >>=1)
   {
      __syncthreads();
      if (threadIdx.x < stride)
      {
         kineticSum[threadIdx.x] += kineticSum[threadIdx.x + stride];
      }
   }
   __syncthreads();
   if (threadIdx.x == 0)
   {   
      kineticArray[blockIdx.x] = kineticSum[0];
   }
}


