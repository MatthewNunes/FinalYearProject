#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define WIDTH 512
#define BLOCK_WIDTH 16

__global__ 
void matMulKernel(float *d_M, float *d_N, float *d_P, int Width)
{ 
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    int k;
    if ((Row<Width)&&(Col<Width)){
        float Pvalue = 0.0;
        for(k=0;k<Width;k++)
        {
            Pvalue += d_M[Row*Width+k]*d_N[k*Width+Col];
        }
        d_P[Row*Width+Col] = Pvalue;
    }
}

void matMulDevice(float *h_M, float *h_N, float *h_P, int Width, float *d_check)
{
    struct timeval tim;
    int size = Width * Width * sizeof(float); 
    float *d_M, *d_N, *d_P;
// Step 1: Allocate and Load M, N to device memory 
    hipMalloc((void **)&d_M, size);
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_N, size);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
// Step 2: Allocate P on the device
    hipMalloc((void **)&d_P, size);
// Step 3a: Set up execution configuration
   int numBlocks = ceil(Width/(float)BLOCK_WIDTH);
   dim3 dimGrid(numBlocks,numBlocks);
   dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
// Step 3b: Launch the device computation threads!
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  // gettimeofday(&tim, NULL);
   hipEventRecord(start, 0);
   //double t1=tim.tv_sec+(tim.tv_usec/1000000.0);
   matMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, Width);
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   float elapsedTime;
   
   hipEventElapsedTime(&elapsedTime, start, stop);
   elapsedTime = elapsedTime / (float) 1000000;
   //gettimeofday(&tim, NULL);
   //double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
// Step 4: Copy back result, and free memory on device
   hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
   int ci = 0;
   int cj = 0;
   int check = 1;
   printf("First Element: %.0lf = %.0lf\n", *h_P, *d_check);
   int count = 0;
   for (ci = 0; ci < Width; ci++)
   {
       for (cj = 0; cj < Width; cj++)
       {
           if (*(h_P +ci * Width + cj) == *(d_check +ci * Width + cj))
           {
              // printf("%.6lf, %.6lf\n", *(h_P +ci * Width + cj), *(d_check +ci * Width + cj));
           }
           else
           {
               count += 1;
               printf("%.6lf, %.6lf\n", *(h_P +ci * Width + cj), *(d_check +ci * Width + cj));
               check = 0;
           }
       }
   }
   printf("Count: %d\n", count);
   hipFree(d_M); hipFree(d_N); hipFree(d_P);
   printf("Check: %d\n", check);
   printf("%.6f seconds elapsed\n", elapsedTime);
}

int main()
{
    float *h_M, *h_N, *h_P, *check;
    int i, n = WIDTH, size=sizeof(float)*n*n;
    h_P = (float *)malloc(size);
    h_M = (float *)malloc(size);
    h_N = (float *)malloc(size);
    check = (float *)malloc(size);
    for(i=0;i<n*n;i++)
    {
        *(h_M+i)=(float)i; 
        *(h_N+i)=(float)i;
    }
    int j;
    int k;
    float *h_MTemp = h_M;
    float *h_NTemp = h_N;
    float sum = 0.0;
    for (i = 0; i < n; i++)
    {
         
        for (j = 0; j < n; j++)
        {
            sum = 0.0;
            for (k =0; k < n; k++)
            {
                sum += h_MTemp[i * n + k] * h_NTemp[k*n + j];
            }
            check[i * n + j] = sum;
        }
        
    }
    /**
    for (i = 0; i < n; i++)
    {
        for (j =0; j + 8 < n; j+=8)
        {
            printf("%f, %f, %f, %f, %f, %f, %f, %f\n", check[i*n + j], check[i*n + j + 1], check[i*n + j + 2], check[i*n + j + 3], check[i*n + j + 4], check[i*n + j + 5], check[i*n + j + 6], check[i*n + j + 7]); 
        }
        printf("%f, %f, %f, %f, %f, %f, %f, %f\n", check[i*n + j], check[i*n + j + 1], check[i*n + j + 2], check[i*n + j + 3], check[i*n + j + 4], check[i*n + j + 5], check[i*n + j + 6], check[i*n + j + 7]);
    }
    */
    matMulDevice(h_M,h_N,h_P,n, check);
}
